#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <set>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <chrono>

#include <opencv2/objdetect/objdetect.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/opencv.hpp"
#include <opencv2/core/core.hpp>

__global__ void helloCUDA()
{
    printf("Hello CUDA1 from GPU!\n");
}

int main()
{
    // 调用CUDA核函数
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    std::cout << "Hello CUDA from CPU!" << std::endl;

    return 0;
}