
#include <hip/hip_runtime.h>
#include <iostream>
/* __global__ 是 CUDA C/C++ 中的一个修饰符，用于声明一个函数为 GPU 的全局函数。在
 * CUDA 编程中，我们可以使用这个修饰符来定义在 GPU 上执行的函数。
 * 具体地说，__global__ 修饰符告诉编译器该函数将在 GPU 上执行，并且可以从主机
 * （CPU）代码中调用。这样的函数被称为 CUDA 内核函数（kernel function），它在每个线程
 * 块（thread block）上并行执行。
 * 在 CUDA 内核函数中，我们可以使用 CUDA 提供的线程和内存管理机制来控制并发执行和数
 * 据传输。在执行 CUDA 内核函数时，我们需要使用特殊的语法来指定线程块和线程的数量，
 * 以便在 GPU 上并行执行代码。
 * 总结起来，__global__ 修饰符用于声明一个在 GPU 上执行的 CUDA 内核函数，它是
 * CUDA 编程中非常重要的概念之一。
 * */
__global__ void helloCUDA()
{
    printf("Hello CUDA0 from GPU!\n");
}

int main()
{
    // 调用CUDA核函数
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    std::cout << "Hello CUDA from CPU!" << std::endl;

    return 0;
}